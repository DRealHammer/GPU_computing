// Sven & Daniel

#include <stdio.h>
#include <stdlib.h>

#include "chTimer.h"


int main(int argc, char** argv) {

    int cIterations = 100000;
    chTimerTimestamp start, stop;

    // memsize in kB
    int memsize = atoi(argv[1]);

    // 1 -> host 2 device, 0 -> device 2 host
    int h2d = atoi(argv[2]);

    int pinned = atoi(argv[3]);

    void* memory;

    // get memory
    if (!pinned) {
        memory = malloc(1000 * memsize);
    } else {
        hipHostMalloc(&memory, 1000 * memsize);
    }
    
    void* d_memory;
    hipMalloc(&d_memory, 1000 * memsize);

    chTimerGetTime( &start );

    if (h2d) {
        for (int i = 0; i < cIterations; i++) {
            hipMemcpy(d_memory, memory, 1000 * memsize, hipMemcpyHostToDevice);
        }
    } else {
        for (int i = 0; i < cIterations; i++) {
            hipMemcpy(memory, d_memory, 1000 * memsize, hipMemcpyDeviceToHost);
        }
    }
    
    chTimerGetTime( &stop );


    {
        double microseconds = 1e6*chTimerElapsedTime( &start, &stop );
        double usPerLaunch = microseconds / (float) cIterations;

        printf( "%d,%d,%d,%.2f\n", memsize, h2d, pinned, usPerLaunch);
    }  

    if (!pinned) {
        free(memory);
    } else {
        hipHostFree(memory);
    }

    hipFree(d_memory);

    

    return 0;


}