/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__device__ clock_t device_clock;

__global__ void BusyWaitKernel(clock_t numClocks) 
{
    auto start = clock64();
    auto end = clock64();

    while (end - start < numClocks) 
    {
        end = clock64();
        
        //Avoid compiler optimization
        if(threadIdx.x == 10)
        {
            device_clock = end - start;
        }
    }
}

int main()
{
    constexpr std::size_t cIterations = 3000;
    constexpr std::size_t runs = 1000;

    //Warm-up
    for( int i = 0; i < 100; ++i ) 
    {
        BusyWaitKernel<<<1, 1>>>(10);
    } 
    hipDeviceSynchronize();

    for( int cycles = 0; cycles <= cIterations; cycles += 100 ) 
    {
        auto start = std::chrono::steady_clock::now();

        for( int j = 0; j < runs; ++j ) 
        {
            BusyWaitKernel<<<1, 1>>>(cycles);
        }

        hipDeviceSynchronize();
        auto end = std::chrono::steady_clock::now();

        std::cout << cycles << " " << (double) std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / runs << std::endl;
    }

    return 0;
}
