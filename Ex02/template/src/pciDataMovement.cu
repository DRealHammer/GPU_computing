/*
 *
 * nullKernelAsync.cu
 *
 * Microbenchmark for throughput of asynchronous kernel launch.
 *
 * Build with: nvcc -I ../chLib <options> nullKernelAsync.cu
 * Requires: No minimum SM requirement.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions 
 * are met: 
 *
 * 1. Redistributions of source code must retain the above copyright 
 *    notice, this list of conditions and the following disclaimer. 
 * 2. Redistributions in binary form must reproduce the above copyright 
 *    notice, this list of conditions and the following disclaimer in 
 *    the documentation and/or other materials provided with the 
 *    distribution. 
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS 
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE 
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER 
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT 
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE 
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main()
{
    constexpr std::size_t min = 1<<10;
    constexpr std::size_t max = 1<<30;

    constexpr std::size_t measurements = 3;

    void* dmem; 
    void* hmem = malloc(max);
    void* hmemPinned;

    //Allocate only once
    hipMalloc(&dmem, max);
    hipHostMalloc(&hmemPinned, max, hipHostMallocDefault);

    //Warmup
    for(int i = 0; i < 2; ++i)
    {
        hipMemcpy(dmem, hmem, 1<<30, hipMemcpyHostToDevice);
        hipMemcpy(hmem, dmem, 1<<30, hipMemcpyDeviceToHost);    
    }

    for(std::size_t i = 0; i < 4; ++i)
    {
        switch(i)
        {
            case 0: std::cout << "copyPageableMemoryHostToDevice (size|time[us]):" << std::endl; break;
            case 1: std::cout << "copyPageableMemoryDeviceToHost (size|time[us]):" << std::endl; break;
            case 2: std::cout << "copyPinnedMemoryHostToDevice (size|time[us]):" << std::endl; break;
            case 3: std::cout << "copyPinnedMemoryDeviceToHost (size|time[us]):" << std::endl; break;
        }

        for(std::size_t size = min; size <= max; size *= 2)
        {
            std::chrono::microseconds duration;

            for(std::size_t k = 0; k < measurements; ++k)
            {
                auto start = std::chrono::steady_clock::now();

                switch(i)
                {
                    case 0: hipMemcpy(dmem, hmem, size, hipMemcpyHostToDevice); break;
                    case 1: hipMemcpy(hmem, dmem, size, hipMemcpyDeviceToHost); break;
                    case 2: hipMemcpy(dmem, hmemPinned, size, hipMemcpyHostToDevice); break;
                    case 3: hipMemcpy(hmemPinned, dmem, size, hipMemcpyDeviceToHost); break;
                }

                auto end = std::chrono::steady_clock::now();

                duration += std::chrono::duration_cast<std::chrono::microseconds>(end - start);
            }

            std::cout << size << " " << (double) duration.count() / measurements << std::endl;

            duration = std::chrono::microseconds();
        }

        std::cout << std::endl;
    }

    hipFree(dmem);
    free(hmem);
    hipHostFree(hmemPinned);

    return 0;
}
