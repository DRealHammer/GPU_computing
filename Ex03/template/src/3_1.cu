
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

int main()
{
    constexpr std::size_t min = 1<<10;
    constexpr std::size_t max = 1<<30;

    constexpr std::size_t measurements = 5;

    void* dmemA; 
    void* dmemB; 
    void* hmem;
    void* hmemPinned;

    //Allocate only once
    hmem = malloc(max);
    hipMalloc(&dmemA, max);
    hipMalloc(&dmemB, max);
    hipHostMalloc(&hmemPinned, max, hipHostMallocDefault);

    //Warmup
    for(int i = 0; i < 2; ++i)
    {
        hipMemcpy(dmemA, hmem, max, hipMemcpyHostToDevice);
        hipMemcpy(hmem, dmemA, max, hipMemcpyDeviceToHost);  
        hipMemcpy(dmemA, hmemPinned, max, hipMemcpyHostToDevice);
        hipMemcpy(hmemPinned, dmemA, max, hipMemcpyDeviceToHost);   
        hipMemcpy(dmemB, dmemA, max, hipMemcpyDeviceToDevice); hipDeviceSynchronize();
    }

    for(std::size_t i = 0; i < 5; ++i)
    {
        switch(i)
        {
            case 0: std::cout << "copyPageableMemoryHostToDevice (size|bandwidth[GB/s]):" << std::endl; break;
            case 1: std::cout << "copyPageableMemoryDeviceToHost (size|bandwidth[GB/s]):" << std::endl; break;
            case 2: std::cout << "copyPinnedMemoryHostToDevice (size|bandwidth[GB/s]):" << std::endl; break;
            case 3: std::cout << "copyPinnedMemoryDeviceToHost (size|bandwidth[GB/s]):" << std::endl; break;
            case 4: std::cout << "copyDeviceToDevice (size|bandwidth[GB/s]):" << std::endl; break;
        }

        for(std::size_t size = min; size <= max; size *= 2)
        {
            std::chrono::microseconds duration;

            for(std::size_t k = 0; k < measurements; ++k)
            {
                auto start = std::chrono::steady_clock::now();

                switch(i)
                {
                    case 0: hipMemcpy(dmemA, hmem, size, hipMemcpyHostToDevice); break;
                    case 1: hipMemcpy(hmem, dmemA, size, hipMemcpyDeviceToHost); break;
                    case 2: hipMemcpy(dmemA, hmemPinned, size, hipMemcpyHostToDevice); break;
                    case 3: hipMemcpy(hmemPinned, dmemA, size, hipMemcpyDeviceToHost); break;
                    case 4: hipMemcpy(dmemB, dmemA, size, hipMemcpyDeviceToDevice); hipDeviceSynchronize(); break;
                }

                auto end = std::chrono::steady_clock::now();

                duration += std::chrono::duration_cast<std::chrono::microseconds>(end - start);
            }

            std::cout << size << " " << ((1e6*static_cast<double>(size) / duration.count()) * measurements) * 1e-9 << std::endl;

            duration = std::chrono::microseconds();
        }

        std::cout << std::endl;
    }

    hipFree(dmemA);
    hipFree(dmemB);
    free(hmem);
    hipHostFree(hmemPinned);

    return 0;
}
