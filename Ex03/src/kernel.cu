
#include <hip/hip_runtime.h>
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel(int memsize_per_thread, int* memA, int* memB)
{
    // number of previous blocks threads + our thread number, accumulated offset
    //int offset = (blockIdx.x * blockDim.x + threadIdx.x ) * memsize_per_thread;
    //void* addr_source = (char*) (memA) + offset;
    //void* addr_target = (char*) (memB) + offset;
    //memcpy(addr_target, addr_source, memsize_per_thread);


    int entries_per_thread = memsize_per_thread / sizeof(int);

    int entry_offset = (blockIdx.x * blockDim.x + threadIdx.x ) * entries_per_thread;

    for (int i = 0; i < entries_per_thread; i++) {
        memB[i + entry_offset] = memA[i + entry_offset];
    }

    
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim, int memsize, int* memA, int* memB) {

    int mem_per_block = memsize / gridDim.x;
    int mem_per_thread = mem_per_block / blockDim.x;
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>(mem_per_thread, memA, memB);
}

__global__ void 
globalMemStrideKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemStrideKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

__global__ void 
globalMemOffsetKernel(/*TODO Parameters*/)
{
    /*TODO Kernel Code*/
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim /*TODO Parameters*/) {
	globalMemOffsetKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>( /*TODO Parameters*/);
}

