
#include <hip/hip_runtime.h>
/*************************************************************************************************
 *
 *        Computer Engineering Group, Heidelberg University - GPU Computing Exercise 03
 *
 *                           Group : TBD
 *
 *                            File : main.cu
 *
 *                         Purpose : Memory Operations Benchmark
 *
 *************************************************************************************************/

//
// Kernels
//

__global__ void 
globalMemCoalescedKernel(/*int memsize_per_thread, */int* memA, int* memB)
{
    // number of previous blocks threads + our thread number, accumulated offset
    //int offset = (blockIdx.x * blockDim.x + threadIdx.x ) * memsize_per_thread;
    //void* addr_source = (char*) (memA) + offset;
    //void* addr_target = (char*) (memB) + offset;
    //memcpy(addr_target, addr_source, memsize_per_thread);


    //int entries_per_thread = memsize_per_thread / sizeof(int);

    //int entry_offset = (blockIdx.x * blockDim.x + threadIdx.x ) * entries_per_thread;

    //memB[entry_offset] = memA[entry_offset];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    memB[tid] = memA[tid];

    
}

void 
globalMemCoalescedKernel_Wrapper(dim3 gridDim, dim3 blockDim, int memsize, int* memA, int* memB) {

    //int mem_per_block = memsize / gridDim.x;
    //int mem_per_thread = mem_per_block / blockDim.x;
	globalMemCoalescedKernel<<< gridDim, blockDim, 0 /*Shared Memory Size*/ >>>(/*mem_per_thread, */memA, memB);
}

__global__ void 
globalMemStrideKernel(int N,int* d_in, int* d_out, int optStride)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;

   if (tid >= N) return;
 
   d_out[tid] = d_in[tid * optStride];
    
}

void 
globalMemStrideKernel_Wrapper(dim3 gridDim, dim3 blockDim, int mem_size, int* d_in, int* d_out, int optStride) {
	
   
    //int threads_per_block;
    //int block_count;
    //int size_per_element;
    //each thread transfers one element
    
    globalMemStrideKernel<<< gridDim, blockDim, 0 >>>(mem_size, d_in, d_out, optStride);
}

__global__ void 
globalMemOffsetKernel(int* d_in, int* d_out, int dataCount, int optOffset)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    //global thread index < available data?
    if(tid < dataCount)
    {
        d_out[tid] = d_in[tid + optOffset];
    }
}

void 
globalMemOffsetKernel_Wrapper(dim3 gridDim, dim3 blockDim, int* d_in, int* d_out,int dataCount, int optOffset) 
{
    globalMemOffsetKernel<<< gridDim, blockDim, 0 >>>(d_in, d_out, dataCount, optOffset);
}

