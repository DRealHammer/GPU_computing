
#include <hip/hip_runtime.h>
/******************************************************************************
 *
 *Computer Engineering Group, Heidelberg University - GPU Computing Exercise 04
 *
 *                  Group : TBD
 *
 *                   File : kernel.cu
 *
 *                Purpose : Memory Operations Benchmark
 *
 ******************************************************************************/


//
// Test Kernel
//

__global__ void  
globalMem2SharedMem(float* gMem, int dataCount)
{
	extern __shared__ float sMem[];

	int currElement = blockIdx.x * blockDim.x + threadIdx.x;

    while(currElement < dataCount) 
	{
        sMem[currElement] = gMem[currElement];
        currElement += blockDim.x * gridDim.x;
    }
}

void globalMem2SharedMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* gMem) 
{
	int dataCount = shmSize / sizeof(float);
	globalMem2SharedMem<<< gridSize, blockSize, shmSize >>>(gMem, dataCount);
}

__global__ void 
SharedMem2globalMem(float* gMem, int dataCount)
{
	extern __shared__ float sMem[];

	int currElement = blockIdx.x * blockDim.x + threadIdx.x;

    while(currElement < dataCount) 
	{
        gMem[currElement] = sMem[currElement];
        currElement += blockDim.x * gridDim.x;
    }
}

void SharedMem2globalMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* gMem) 
{
	int dataCount = shmSize / sizeof(float);
	SharedMem2globalMem<<< gridSize, blockSize, shmSize >>>(gMem, dataCount);
}

__global__ void 
SharedMem2Registers(float* out, int dataCount)
{
	extern __shared__ float sMem[];

	const int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int currElement = tid;
	float reg;

    while(currElement < dataCount) 
	{
        reg = sMem[currElement];
        currElement += blockDim.x * gridDim.x;
    }

	// Prevent compiler optimizations
	if(tid == 0)
	{
		*out = reg;
	}
}

void SharedMem2Registers_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize, float* out) 
{
	int dataCount = shmSize / sizeof(float);
	SharedMem2Registers<<< gridSize, blockSize, shmSize >>>(out, dataCount);
}

__global__ void 
Registers2SharedMem(int dataCount)
{
	extern __shared__ float sMem[];

	int currElement = blockIdx.x * blockDim.x + threadIdx.x;

    while(currElement < dataCount) 
	{
        sMem[currElement] = currElement;
        currElement += blockDim.x * gridDim.x;
    }
}

void Registers2SharedMem_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize) 
{
	int dataCount = shmSize / sizeof(float);
	Registers2SharedMem<<< gridSize, blockSize, shmSize >>>(dataCount);
}

__global__ void 
bankConflictsRead
//(/*TODO Parameters*/)
( )
{
	/*TODO Kernel Code*/
}

void bankConflictsRead_Wrapper(dim3 gridSize, dim3 blockSize, int shmSize /* TODO Parameters*/) {
	bankConflictsRead<<< gridSize, blockSize, shmSize >>>( /* TODO Parameters */);
}
