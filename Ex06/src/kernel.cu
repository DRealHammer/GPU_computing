#include "hip/hip_runtime.h"
/**************************************************************************************************
 *
 *       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
 *
 *                  Group : 01
 *
 *                   File : kernel.cu
 *
 *                Purpose : Reduction
 *
 **************************************************************************************************/

#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

//
// Reduction_Kernel
//
__global__ void reduction_Kernel_6_3(int numElements, float* dataIn, float* dataOut)
{
	extern __shared__ float sPartials[];
	const std::size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid < numElements)
	{
		//Load all elements from global memory into shared memory
		sPartials[threadIdx.x] = dataIn[tid];
		for(std::size_t i = tid + gridDim.x * blockDim.x; i < numElements; i += (gridDim.x * blockDim.x))
		{
			sPartials[threadIdx.x] += dataIn[i]; 
		}

		__syncthreads();

		for(std::size_t s = 1; s < blockDim.x; s *= 2) 
		{
			if(threadIdx.x % (2 * s) == 0) 
			{
				sPartials[threadIdx.x] += sPartials[threadIdx.x + s];
			}

			__syncthreads();
		}
			
		if(threadIdx.x == 0) 
			dataOut[blockIdx.x] = sPartials[0];
	}
}

__global__ void reduction_Kernel_6_4(int numElements, float* dataIn, float* dataOut)
{
	extern __shared__ float sPartials[];
	const std::size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid < numElements)
	{
		//Load all elements from global memory into shared memory
		sPartials[threadIdx.x] = dataIn[tid];
		for(std::size_t i = tid + gridDim.x * blockDim.x; i < numElements; i += (gridDim.x * blockDim.x))
		{
			sPartials[threadIdx.x] += dataIn[i]; 
		}

		__syncthreads();

		//std::size_t i = blockIdx.x * (blockDim.x * 2) + threadIdx.x; //REDUCTION #4: FIRST ADD DURING LOAD
		//sPartials[tid] = dataIn[i] + dataIn[i + blockDim.x]; //REDUCTION #4: FIRST ADD DURING LOAD
		// => Reduction#4: ***ERROR*** 700 - an illegal memory access was encountered***[0m[31m***
		//for(std::size_t s = 1; s < blockDim.x; s *= 2)
		for(std::size_t o = blockDim.x / 2; o > 0 ; o >>= 1) //REDUCTION #3: SEQUENTIAL ADDRESSING NONDIVERGENT
		{//REDUCTION #5: UNROLLING THE LAST WARP (o > 32)
			//int index = 2 * s * tid; //REDUCTION #2: INTERLEAVED ADDRESSING NONDIVERGENT
			//if(threadIdx.x % (2 * s) == 0) // REDUCTION #1: INTERLEAVED ADDRESSING was already done
			//if (index < blockDim.x)
			if(tid < o)
			{
				//sPartials[index] += sPartials[index + s]; //REDUCTION #2: INTERLEAVED ADDRESSING NONDIVERGENT
				sPartials[tid] += sPartials[tid + o]; //REDUCTION #3: SEQUENTIAL ADDRESSING NONDIVERGENT
			}

			__syncthreads();
		}

		//REDUCTION #5: UNROLLING THE LAST WARP
		if ( tid < 32 && blockDim.x >= 64) sPartials[tid] += sPartials[tid + 32];
		if ( tid < 16 && blockDim.x >= 32) sPartials[tid] += sPartials[tid + 16];
		if ( tid < 8 && blockDim.x >= 16) sPartials[tid] += sPartials[tid + 8];
		if ( tid < 4 && blockDim.x >= 8) sPartials[tid] += sPartials[tid + 4];
		if ( tid < 2 && blockDim.x >= 4) sPartials[tid] += sPartials[tid + 2];
		if ( tid < 1 && blockDim.x >= 2) sPartials[tid] += sPartials[tid + 1]; 

			
		if(threadIdx.x == 0) 
			dataOut[blockIdx.x] = sPartials[0];
	}
}

__inline__ __device__ float warpReduceSum(float val) 
{
	//Calculate the sum of all elements within one warp (32 elements) 
	for(std::size_t offset = warpSize / 2; offset > 0; offset >>= 1) 
		val += __shfl_down_sync(0xFFFFFFFF, val, offset);

	return val;
}

__inline__ __device__ float blockReduceSum(float val)
{
	static __shared__ float sPartials[32];	//warpSize
	const std::size_t lane = threadIdx.x % warpSize;
	const std::size_t wid = threadIdx.x / warpSize;

	//Each warp performs partial reduction
	val = warpReduceSum(val);     	

	//First thread of each warp writes its partial sum to shared memory
	if(lane == 0) 
		sPartials[wid] = val; 		

	//Wait for all partial reductions
	__syncthreads();              	

	//Read from shared memory only if that warp existed
	val = (threadIdx.x < blockDim.x / warpSize + (blockDim.x % 32 != 0)) ? sPartials[lane] : 0;

	if(wid == 0) 
		val = warpReduceSum(val); 	//Final reduce within first warp

	return val;
}

__global__ void reduction_Kernel_6_5(int numElements, float* dataIn, float* dataOut)
{
	const std::size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < numElements)
	{
		int sum = dataIn[tid];
		//Reduce all elements which are not covered by a thread
		for(std::size_t i = tid + gridDim.x * blockDim.x; i < numElements; i += (gridDim.x * blockDim.x))
		{
			sum += dataIn[i]; 
		}

		sum = blockReduceSum(sum);

		if(threadIdx.x == 0) 
			dataOut[blockIdx.x] = sum;
	}
}

void reduction_Kernel_Wrapper_6_3(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) 
{
	reduction_Kernel_6_3<<< gridSize, blockSize, blockSize.x*sizeof(float) /*Shared Mem*/ >>>(numElements, dataIn, dataOut);
}

void reduction_Kernel_Wrapper_6_4(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) 
{
	reduction_Kernel_6_4<<< gridSize, blockSize, blockSize.x*sizeof(float) /*Shared Mem*/ >>>(numElements, dataIn, dataOut);
}

void reduction_Kernel_Wrapper_6_5(dim3 gridSize, dim3 blockSize, int numElements, float* dataIn, float* dataOut) 
{
	reduction_Kernel_6_5<<< gridSize, blockSize>>>(numElements, dataIn, dataOut);
}

//
// Reduction Kernel using CUDA Thrust
//

void thrust_reduction_Wrapper(int numElements, float* dataIn, float* dataOut) 
{
	thrust::device_ptr<float> in_ptr = thrust::device_pointer_cast(dataIn);
	thrust::device_ptr<float> out_ptr = thrust::device_pointer_cast(dataOut);
	
	*out_ptr = thrust::reduce(in_ptr, in_ptr + numElements, (float) 0., thrust::plus<float>());	
}
